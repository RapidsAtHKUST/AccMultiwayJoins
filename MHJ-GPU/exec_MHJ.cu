#include "ooc.cuh"
#include "QueryProcessor.cuh"

template<typename DataType, typename CntType>
void test_general(string dir_name, bool ooc) {
    CUDAMemStat memstat;
    CUDATimeStat timing;
    QueryProcessor<DataType, CntType> processor;
    Relation<DataType,CntType> *relations = nullptr;
    HashTable<DataType,CntType> *hash_tables = nullptr;
    CntType *bucket_vec = nullptr;
    uint32_t num_tables;
    uint32_t gpu_time_idx;
    float build_kernel_time, probe_kernel_time;
    Timer t;

    /*load data from disk*/
    processor.load_multi_rels(dir_name, relations, num_tables, &memstat);
    uint32_t num_hash_tables = num_tables - 1;

    /*analyze the query datasets*/
    int num_attrs_in_res = 0; //number of output attrs
    int num_attr_idxes_in_iRes=0; //number of output attrs in iRes
    AttrType *attr_idxes_in_iRes = nullptr;
    CUDA_MALLOC(&attr_idxes_in_iRes, sizeof(AttrType)*MAX_NUM_RES_ATTRS, &memstat);

    /*compute used_for_compare*/
    bool **used_for_compare = nullptr;
    bool attr_referred[MAX_NUM_RES_ATTRS] = {false};
    CUDA_MALLOC(&used_for_compare, sizeof(bool*)*num_hash_tables, &memstat);
    for(auto i = 0; i < num_tables; i++) { /*compute used_for_compare array*/
        if (0 != i)
            CUDA_MALLOC(&used_for_compare[i-1], sizeof(bool)*relations[i].num_attrs, &memstat);
        for(auto a = 0; a < relations[i].num_attrs; a++) {
            if (!attr_referred[relations[i].attr_list[a]]) {//this attr has not shown in previous relations
                attr_referred[relations[i].attr_list[a]] = true;
                num_attrs_in_res++;
                if (0 != i) used_for_compare[i-1][a] = false;
                if (i != num_tables -1) attr_idxes_in_iRes[num_attr_idxes_in_iRes++] = relations[i].attr_list[a];
            }
            else if (0 != i) used_for_compare[i-1][a] = true;
        }
    }

    /*prefetch relation data*/
    hipDeviceSynchronize();
    t.reset();
    for(auto i = 0; i < num_tables; i++) {
        if ((0 == i) && (relations[i].length > MAX_PROBE_PREFETCH_THRES)) { //do not prefetch the probe table
            log_info("Skip prefetch of table 0");
            continue;
        }
        for(auto a = 0; a < relations[i].num_attrs; a++) {
            checkCudaErrors(hipMemPrefetchAsync(relations[i].data[a], sizeof(DataType)*relations[i].length, DEVICE_ID));
            log_info("Prefetch col %d of table %d", a, i);
        }
    }

    /*build phase*/
    gpu_time_idx = timing.get_idx();
    processor.build_hash_multiway(&relations[1], hash_tables, bucket_vec, num_tables - 1, MHJ_BUC_RATIO, &memstat, &timing);
    build_kernel_time = timing.diff_time(gpu_time_idx);

    size_t free_byte, total_byte;
    auto cuda_status = hipMemGetInfo(&free_byte, &total_byte);//show memory usage of GPU
    if (hipSuccess != cuda_status){
        log_error("hipMemGetInfo fails, %s", hipGetErrorString(cuda_status));
        exit(1);
    }
    auto used_byte = total_byte - free_byte;
    log_info("GPU memory usage: used = %.1f MB, free = %.1f MB, total = %.1f MB",
             1.0*used_byte/1024.0/1024.0, 1.0*free_byte/1024.0/1024.0, 1.0*total_byte/1024.0/1024.0);
    auto total_budget = (bsize_t)free_byte; //double buffer
    log_info("Initialize OOCWrapper with budget: %.1f MB", 1.0*total_budget/1024.0/1024.0);

    OOC<DataType,CntType, false, TYPE_MHJ> ooc_wrapper(1.0*total_budget/1024.0/1024.0);

    /*probe phase*/
    DataType **res_dummy = nullptr;
    gpu_time_idx = timing.get_idx();
    ooc_wrapper.execute(relations[0], hash_tables, num_hash_tables, used_for_compare,
                        bucket_vec, num_attrs_in_res, attr_idxes_in_iRes,
                        num_attr_idxes_in_iRes, ooc, res_dummy, &memstat, &timing);
    probe_kernel_time = timing.diff_time(gpu_time_idx);

    log_info("---------------------------------");
    log_info("Build kernel time: %.2f ms", build_kernel_time);
    log_info("Probe kernel time: %.2f ms", probe_kernel_time);
    log_info("Total kernel time: %.0f ms", build_kernel_time+probe_kernel_time);
    log_info("Total CPU execution time: %.2f ms", t.elapsed()*1000);
    log_info("Maximal device mem demanded: %ld bytes.", memstat.get_max_use());
    log_info("Unfreed mem size: %ld bytes.", memstat.get_cur_use());
}

/*
 * ./cuda-mwtbj DATA_DIR OOC
 * */
int main(int argc, char *argv[]) {
    hipSetDevice(DEVICE_ID);

    if (argc == 4) {
        FILE *fp;
        fp = fopen(argv[3], "a+");
        if (fp == NULL) {
            cout<<"wrong file fp"<<endl;
            exit(1);
        }
        log_set_fp(fp);
    }

    test_general<KeyType,CarType>(string(argv[1]), (bool)stoi(argv[2]));
    return 0;
}