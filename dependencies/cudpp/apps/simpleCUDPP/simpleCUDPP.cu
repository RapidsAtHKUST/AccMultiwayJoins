#include "hip/hip_runtime.h"
// -------------------------------------------------------------
// cuDPP -- CUDA Data Parallel Primitives library
// -------------------------------------------------------------
// $Revision$
// $Date$
// ------------------------------------------------------------- 
// This source code is distributed under the terms of license.txt in
// the root directory of this source distribution.
// ------------------------------------------------------------- 

/*
 * This is a basic example of how to use the CUDPP library.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include "cudpp.h"

#include <string>

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

extern "C" 
void computeSumScanGold( float *reference, const float *idata, 
                        const unsigned int len,
                        const CUDPPConfiguration &config);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
    runTest( argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        fprintf(stderr, "error: no devices supporting CUDA.\n");
        exit(EXIT_FAILURE);
    }
    int dev = 0;
    if (argc > 1) {
        std::string arg = argv[1];
        size_t pos = arg.find("=");
        if (arg.find("device") && pos != std::string::npos) {
            dev = atoi(arg.c_str() + (pos + 1));
        }
    }
    if (dev < 0) dev = 0;
    if (dev > deviceCount-1) dev = deviceCount - 1;
    hipSetDevice(dev);

    hipDeviceProp_t prop;
    if (hipGetDeviceProperties(&prop, dev) == hipSuccess)
    {
        printf("Using device %d:\n", dev);
        printf("%s; global mem: %dB; compute v%d.%d; clock: %d kHz\n",
               prop.name, (int)prop.totalGlobalMem, (int)prop.major, 
               (int)prop.minor, (int)prop.clockRate);
    }

    unsigned int numElements = 32768;
    unsigned int memSize = sizeof( float) * numElements;

    // allocate host memory
    float* h_idata = (float*) malloc( memSize);
    // initalize the memory
    for (unsigned int i = 0; i < numElements; ++i) 
    {
        h_idata[i] = (float) (rand() & 0xf);
    }

    // allocate device memory
    float* d_idata;
    hipError_t result = hipMalloc( (void**) &d_idata, memSize);
    if (result != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(result));
        exit(-1);
    }
    
    // copy host memory to device
    result = hipMemcpy( d_idata, h_idata, memSize, hipMemcpyHostToDevice);
    if (result != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(result));
        exit(-1);
    }
     
    // allocate device memory for result
    float* d_odata;
    result = hipMalloc( (void**) &d_odata, memSize);
    if (result != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(result));
        exit(-1);
    }

    // Initialize the CUDPP Library
    CUDPPHandle theCudpp;
    cudppCreate(&theCudpp);

    CUDPPConfiguration config;
    config.op = CUDPP_ADD;
    config.datatype = CUDPP_FLOAT;
    config.algorithm = CUDPP_SCAN;
    config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE;
    
    CUDPPHandle scanplan = 0;
    CUDPPResult res = cudppPlan(theCudpp, &scanplan, config, numElements, 1, 0);  

    if (CUDPP_SUCCESS != res)
    {
        printf("Error creating CUDPPPlan\n");
        exit(-1);
    }

    // Run the scan
    res = cudppScan(scanplan, d_odata, d_idata, numElements);
    if (CUDPP_SUCCESS != res)
    {
        printf("Error in cudppScan()\n");
        exit(-1);
    }

    // allocate mem for the result on host side
    float* h_odata = (float*) malloc( memSize);
    // copy result from device to host
    result = hipMemcpy( h_odata, d_odata, memSize, hipMemcpyDeviceToHost);
    if (result != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(result));
        exit(-1);
    }
    
    // compute reference solution
    float* reference = (float*) malloc( memSize);
    computeSumScanGold( reference, h_idata, numElements, config);

    // check result
    bool passed = true;
    for (unsigned int i = 0; i < numElements; i++)
        if (reference[i] != h_odata[i]) passed = false;
        
    printf( "Test %s\n", passed ? "PASSED" : "FAILED");

    res = cudppDestroyPlan(scanplan);
    if (CUDPP_SUCCESS != res)
    {
        printf("Error destroying CUDPPPlan\n");
        exit(-1);
    }

    // shut down the CUDPP library
    cudppDestroy(theCudpp);
    
    free( h_idata);
    free( h_odata);
    free( reference);
    hipFree(d_idata);
    hipFree(d_odata);
}
