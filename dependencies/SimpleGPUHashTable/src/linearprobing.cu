#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdint.h"
#include "vector"
#include "linearprobing.h"

// 32 bit Murmur3 hash
__device__ uint32_t hash(uint32_t k)
{
    k ^= k >> 16;
    k *= 0x85ebca6b;
    k ^= k >> 13;
    k *= 0xc2b2ae35;
    k ^= k >> 16;
    return k & (kHashTableCapacity-1);
}

// Create a hash table. For linear probing, this is just an array of KeyValues
KeyValue* create_hashtable() 
{
    // Allocate memory
    KeyValue* hashtable;
    hipMalloc(&hashtable, sizeof(KeyValue) * kHashTableCapacity);

    // Initialize hash table to empty
    static_assert(kEmpty == 0xffffffff, "memset expected kEmpty=0xffffffff");
    hipMemset(hashtable, 0xff, sizeof(KeyValue) * kHashTableCapacity);

    return hashtable;
}

// Insert the key/values in kvs into the hashtable
__global__ void gpu_hashtable_insert(KeyValue* hashtable, const KeyValue* kvs, unsigned int numkvs)
{
    unsigned int threadid = blockIdx.x*blockDim.x + threadIdx.x;
    if (threadid < numkvs)
    {
        uint32_t key = kvs[threadid].key;
        uint32_t value = kvs[threadid].value;
        uint32_t slot = hash(key);

        while (true)
        {
            uint32_t prev = atomicCAS(&hashtable[slot].key, kEmpty, key);
            if (prev == kEmpty || prev == key)
            {
                hashtable[slot].value = value;
                return;
            }

            slot = (slot + 1) & (kHashTableCapacity-1);
        }
    }
}
 
void insert_hashtable(KeyValue* pHashTable, const KeyValue* kvs, uint32_t num_kvs)
{
    // Copy the keyvalues to the GPU
    KeyValue* device_kvs;
    hipMalloc(&device_kvs, sizeof(KeyValue) * num_kvs);
    hipMemcpy(device_kvs, kvs, sizeof(KeyValue) * num_kvs, hipMemcpyHostToDevice);

    // Have CUDA calculate the thread block size
    int mingridsize;
    int threadblocksize;
    hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, gpu_hashtable_insert, 0, 0);

    // Create events for GPU timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Insert all the keys into the hash table
    int gridsize = ((uint32_t)num_kvs + threadblocksize - 1) / threadblocksize;
    gpu_hashtable_insert<<<gridsize, threadblocksize>>>(pHashTable, device_kvs, (uint32_t)num_kvs);

    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    float seconds = milliseconds / 1000.0f;
    printf("    GPU inserted %d items in %f ms (%f million keys/second)\n", 
        num_kvs, milliseconds, num_kvs / (double)seconds / 1000000.0f);

    hipFree(device_kvs);
}

// Lookup keys in the hashtable, and return the values
__global__ void gpu_hashtable_lookup(KeyValue* hashtable, KeyValue* kvs, unsigned int numkvs)
{
    unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadid < kHashTableCapacity)
    {
        uint32_t key = kvs[threadid].key;
        uint32_t slot = hash(key);

        while (true)
        {
            if (hashtable[slot].key == key)
            {
                kvs[threadid].value = hashtable[slot].value;
                return;
            }
            if (hashtable[slot].key == kEmpty)
            {
                kvs[threadid].value = kEmpty;
                return;
            }
            slot = (slot + 1) & (kHashTableCapacity - 1);
        }
    }
}

void lookup_hashtable(KeyValue* pHashTable, KeyValue* kvs, uint32_t num_kvs)
{
    // Copy the keyvalues to the GPU
    KeyValue* device_kvs;
    hipMalloc(&device_kvs, sizeof(KeyValue) * num_kvs);
    hipMemcpy(device_kvs, kvs, sizeof(KeyValue) * num_kvs, hipMemcpyHostToDevice);

    // Have CUDA calculate the thread block size
    int mingridsize;
    int threadblocksize;
    hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, gpu_hashtable_insert, 0, 0);

    // Create events for GPU timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Insert all the keys into the hash table
    int gridsize = ((uint32_t)num_kvs + threadblocksize - 1) / threadblocksize;
    gpu_hashtable_insert << <gridsize, threadblocksize >> > (pHashTable, device_kvs, (uint32_t)num_kvs);

    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    float seconds = milliseconds / 1000.0f;
    printf("    GPU lookup %d items in %f ms (%f million keys/second)\n",
        num_kvs, milliseconds, num_kvs / (double)seconds / 1000000.0f);

    hipFree(device_kvs);
}

// Delete each key in kvs from the hash table, if the key exists
// A deleted key is left in the hash table, but its value is set to kEmpty
// Deleted keys are not reused; once a key is assigned a slot, it never moves
__global__ void gpu_hashtable_delete(KeyValue* hashtable, const KeyValue* kvs, unsigned int numkvs)
{
    unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadid < kHashTableCapacity)
    {
        uint32_t key = kvs[threadid].key;
        uint32_t slot = hash(key);

        while (true)
        {
            if (hashtable[slot].key == key)
            {
                hashtable[slot].value = kEmpty;
                return;
            }
            if (hashtable[slot].key == kEmpty)
            {
                return;
            }
            slot = (slot + 1) & (kHashTableCapacity - 1);
        }
    }
}

void delete_hashtable(KeyValue* pHashTable, const KeyValue* kvs, uint32_t num_kvs)
{
    // Copy the keyvalues to the GPU
    KeyValue* device_kvs;
    hipMalloc(&device_kvs, sizeof(KeyValue) * num_kvs);
    hipMemcpy(device_kvs, kvs, sizeof(KeyValue) * num_kvs, hipMemcpyHostToDevice);

    // Have CUDA calculate the thread block size
    int mingridsize;
    int threadblocksize;
    hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, gpu_hashtable_insert, 0, 0);

    // Create events for GPU timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Insert all the keys into the hash table
    int gridsize = ((uint32_t)num_kvs + threadblocksize - 1) / threadblocksize;
    gpu_hashtable_delete<< <gridsize, threadblocksize >> > (pHashTable, device_kvs, (uint32_t)num_kvs);

    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    float seconds = milliseconds / 1000.0f;
    printf("    GPU delete %d items in %f ms (%f million keys/second)\n",
        num_kvs, milliseconds, num_kvs / (double)seconds / 1000000.0f);

    hipFree(device_kvs);
}

// Iterate over every item in the hashtable; return non-empty key/values
__global__ void gpu_iterate_hashtable(KeyValue* pHashTable, KeyValue* kvs, uint32_t* kvs_size)
{
    unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadid < kHashTableCapacity) 
    {
        if (pHashTable[threadid].key != kEmpty) 
        {
            uint32_t value = pHashTable[threadid].value;
            if (value != kEmpty)
            {
                uint32_t size = atomicAdd(kvs_size, 1);
                kvs[size] = pHashTable[threadid];
            }
        }
    }
}

std::vector<KeyValue> iterate_hashtable(KeyValue* pHashTable)
{
    uint32_t* device_num_kvs;
    hipMalloc(&device_num_kvs, sizeof(uint32_t));
    hipMemset(device_num_kvs, 0, sizeof(uint32_t));

    KeyValue* device_kvs;
    hipMalloc(&device_kvs, sizeof(KeyValue) * kNumKeyValues);

    int mingridsize;
    int threadblocksize;
    hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, gpu_iterate_hashtable, 0, 0);

    int gridsize = (kHashTableCapacity + threadblocksize - 1) / threadblocksize;
    gpu_iterate_hashtable<<<gridsize, threadblocksize>>>(pHashTable, device_kvs, device_num_kvs);

    uint32_t num_kvs;
    hipMemcpy(&num_kvs, device_num_kvs, sizeof(uint32_t), hipMemcpyDeviceToHost);

    std::vector<KeyValue> kvs;
    kvs.resize(num_kvs);

    hipMemcpy(kvs.data(), device_kvs, sizeof(KeyValue) * num_kvs, hipMemcpyDeviceToHost);

    hipFree(device_kvs);
    hipFree(device_num_kvs);

    return kvs;
}

// Free the memory of the hashtable
void destroy_hashtable(KeyValue* pHashTable)
{
    hipFree(pHashTable);
}
